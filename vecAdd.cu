#include "hip/hip_runtime.h"
%%writefile vecAdd.cu
#include <iostream>
#include <hip/hip_runtime.h>
// !nvcc -arch=sm_75 -ccbin=/usr/bin/gcc-11 vecAdd.cu -o vecAdd -lstdc++ && ./vecAdd
using namespace std;
__global__ void vecAdd(int* a,int* b,int*c,int n){
    int tid=blockIdx.x*blockDim.x+threadIdx.x;
    if(tid<n){
	c[tid]=a[tid]+b[tid];
    }
}
int main(){
    int N=5;
    int A[N]={1,2,3,4,5};
    int B[N]={2,3,4,5,6};
    int C[N];
    int totalsize=N*sizeof(int);
    int *X,*Y,*Z;
    hipMalloc(&X,totalsize);
    hipMalloc(&Y,totalsize);
    hipMalloc(&Z,totalsize);
    
    hipMemcpy(X,A,totalsize,hipMemcpyHostToDevice);
    hipMemcpy(Y,B,totalsize,hipMemcpyHostToDevice);
    vecAdd<<<1,N>>>(X,Y,Z,N);
    hipDeviceSynchronize();
    hipMemcpy(C,Z,totalsize,hipMemcpyDeviceToHost);
    cout<<"A ";
    for(int i=0;i<N;i++) cout<<A[i]<<" ";
    cout<<endl;
    cout<<"B ";
    for(int i=0;i<N;i++) cout<<B[i]<<" ";
    cout<<endl;
    cout<<"C ";
    for(int i=0;i<N;i++) cout<<C[i]<<" ";
    cout<<endl;
}
