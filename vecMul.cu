#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;
__global__ void vecMul(int*a,int*b,int* c,int n){
    int row=blockIdx.y*blockDim.y+threadIdx.y;
    int col=blockIdx.x*blockDim.x+threadIdx.x;
    int sum=0;
    for(int k=0;k<n;++k){
	sum+=(a[row*n+k]*b[k*n+col]);
    }
    c[row*n+col]=sum;
}
int main(){
    int N=2;
    int A[4]={1,2,3,4};
    int B[4]={2,3,4,5};
    int C[4];
    int *X,*Y,*Z;
    int totalsize=4*sizeof(int);
    hipMalloc(&X,totalsize);
    hipMalloc(&Y,totalsize);
    hipMalloc(&Z,totalsize);

    hipMemcpy(X,A,totalsize,hipMemcpyHostToDevice);
    hipMemcpy(Y,B,totalsize,hipMemcpyHostToDevice);

    dim3 threadsPerBlock2D(2,2);
    dim3 blocksPerGrid2D(1,1);
    vecMul<<<blocksPerGrid2D,threadsPerBlock2D>>>(X,Y,Z,N);
    hipMemcpy(C,Z,totalsize,hipMemcpyDeviceToHost);
   
    cout << "Matrix A:\n";
    cout << A[0] << " " << A[1] << "\n" << A[2] << " " << A[3] << "\n";

    cout << "Matrix B:\n";
    cout << B[0] << " " << B[1] << "\n" << B[2] << " " << B[3] << "\n";

    cout << "Matrix C = A x B:\n";
    cout << C[0] << " " << C[1] << "\n" << C[2] << " " << C[3] << "\n";
}
